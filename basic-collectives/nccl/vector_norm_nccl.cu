#include "hip/hip_runtime.h"
#include <stdio.h>
#include "nccl.h"
#include <stdlib.h>
/* #include <iostream> */
#include <stdexcept>
#include <type_traits>
#include "hipblas.h"
#include <hip/hip_runtime.h>
#include "dot.cuh"

template <typename T>
void check_arrays(int sz, T** sendbuff);

template <typename T>
void check_arrays(int sz, T** sendbuff){
  T* host_arr = (T*)malloc(sz * sizeof(T));
  /* hipSetDevice(0); */
  T host_val = -999.;
  printf("norm val %f\n", host_val);
  hipMemcpy(&host_val, &sendbuff[0][0], sizeof(T), hipMemcpyDeviceToHost);
  printf("host single value  : %f \n", host_val);
  hipMemcpy(host_arr, sendbuff[0], sz*sizeof(T), hipMemcpyDeviceToHost);
  printf("host arr value  : %f \n", host_arr[0]);

  /* for (int i = 0; i < sz; i++){ */
  /*   std::cout << host_arr[i] << std::endl; */
  /* } */ 
}

template <typename T>
void test_init(){
  int nDev = 2;
  int sz = 32;

  T* host_v = (T*)malloc(sz * sizeof(T));
  for (int i = 0; i < sz; i++){
    host_v[i] = 10.0;
  }

  T** sendbuff = (T**)malloc(nDev * sizeof(T*));

  T* devicebuff;
  /* for (int i = 0; i < nDev; ++i) { */
  /*   hipSetDevice(i); */
  hipMalloc(&devicebuff, sz * sizeof(T));
  hipMemcpy((void*)devicebuff, (const void*)host_v, sz, hipMemcpyHostToDevice); 

  T val = -999;
  hipMemcpy(&val, &devicebuff[0], sizeof(T), hipMemcpyDeviceToHost);
  printf("norm val %f\n", val);
  T* host_arr = (T*)malloc(sz * sizeof(T));
  hipMemcpy(host_arr, devicebuff, sz*sizeof(T), hipMemcpyDeviceToHost);
  printf("norm arr %f", host_arr[0]);
  /* } */
}

template <class T>
struct dependent_false : std::false_type
{
};


/* template <typename T> */
/* T dot_cublas(hipblasHandle_t handle, std::size_t n, T* x, T* y) */
/* { */
/*   T result = 0; */
/*   if constexpr (std::is_same<T, double>()) */
/*     hipblasDdot(handle, n, x, 1, y, 1, &result); */
/*   else if constexpr (std::is_same<T, float>()) */
/*     hipblasSdot(handle, n, x, 1, y, 1, &result); */
/*   else */
/*     static_assert(dependent_false<T>::value); */

/*   return result; */
/* } */

// based on example 1:
// https://docs.nvidia.com/deeplearning/nccl/user-guide/docs/examples.html
// Example 1: Single Process, Single Thread, Multiple Devices
template <typename T>
void vector_norm_ssm(){
  int nDev = 2;
  int sz = 32;

  int devs[2] = {0, 1};
  /* hipblasHandle_t handle; */
  /* hipblasCreate(&handle); */
  T value = 10.;
  T* host_v = (T*)malloc(sz * sizeof(T));
  for (int i = 0; i < sz; i++){
    host_v[i] = value;
  }

  T** sendbuff = (T**)malloc(nDev * sizeof(T*));
  T* recvbuff;
  T* valbuff;
  
  hipMalloc(&recvbuff, nDev * sizeof(T));
  hipMalloc(&valbuff, nDev * sizeof(T));

  ncclComm_t comms[nDev];
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);

  for (int i = 0; i < nDev; ++i) {
    hipSetDevice(i);
    hipMalloc(sendbuff + i, sz * sizeof(T));
    hipMemcpy(sendbuff[i], host_v, sz * sizeof(T), hipMemcpyHostToDevice); /* T float_val = 1.0; */ 
    hipStreamCreate(s+i);
  }

  check_arrays(sz, sendbuff);

  ncclCommInitAll(comms, nDev, devs);

  ncclGroupStart();
  /* hipblasHandle_t handle; */
  /* hipblasCreate(&handle); */
  for (int i = 0; i < nDev; ++i){
    /* T value = dot(handle, sz * sizeof(T), sendbuff[i], sendbuff[i]); */
    dot<<<1,32>>>(sz,  sendbuff[i], sendbuff[i], recvbuff);
    ncclAllReduce(recvbuff, valbuff, sz * sizeof(T), ncclFloat, ncclSum, comms[i], s[i]);
  }
  ncclGroupEnd();

  T val = -999.;
  hipMemcpy(&val, &valbuff[0], sizeof(T), hipMemcpyDeviceToHost);
  printf("single value %f", val);

}



int main()
{
  test_init_2<float>();

}

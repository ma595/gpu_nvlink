#include "hip/hip_runtime.h"
#include <stdio.h>
#include "nccl.h"
#include <stdlib.h>
#include <iostream>
#include <stdexcept>
/* #include <constexpr> */
#include <stdexcept>
#include "hipblas.h"
#include <hip/hip_runtime.h>
#include <type_traits>
#include "dot.cuh"

template <class T>
struct dependent_false : std::false_type
{
};

/* template <typename T> */
/* T dot_cublas(hipblasHandle_t handle, std::size_t n, T* x, T* y) */
/* { */
/*   T result = 0; */
/*   if constexpr (std::is_same<T, double>()) */
/*     hipblasDdot(handle, n, x, 1, y, 1, &result); */
/*   else if constexpr (std::is_same<T, float>()) */
/*     hipblasSdot(handle, n, x, 1, y, 1, &result); */
/*   else */
/*     static_assert(dependent_false<T>::value); */

/*   return result; */
/* } */

template <typename T>
void check_arrays(int sz, T** vec);

// check arrays on device
template <typename T>
void check_arrays(int sz, T** vec){
  T* host_arr = (T*)malloc(sz * sizeof(T));
  /* hipSetDevice(0); */
  T host_val = -999.;
  printf("norm val %f\n", host_val);
  hipMemcpy(&host_val, &vec[0][0], sizeof(T), hipMemcpyDeviceToHost);
  printf("host single value  : %f \n", host_val);
  hipMemcpy(host_arr, vec[0], sz*sizeof(T), hipMemcpyDeviceToHost);
  printf("host arr value  : %f \n", host_arr[0]);

  for (int i = 0; i < sz; i++){
    std::cout << host_arr[i] << std::endl;
  } 
}

// check cuda functions
template <typename T>
void test_init(){
  int nDev = 2;
  int sz = 32;

  T* host_v = (T*)malloc(sz * sizeof(T));
  for (int i = 0; i < sz; i++){
    host_v[i] = 10.0;
  }

  T** vec = (T**)malloc(nDev * sizeof(T*));

  T* devicebuff;
  /* for (int i = 0; i < nDev; ++i) { */
  /*   hipSetDevice(i); */
  hipMalloc(&devicebuff, sz * sizeof(T));
  hipMemcpy((void*)devicebuff, (const void*)host_v, sz, hipMemcpyHostToDevice); 

  T val = -999;
  hipMemcpy(&val, &devicebuff[0], sizeof(T), hipMemcpyDeviceToHost);
  printf("norm val %f\n", val);
  T* host_arr = (T*)malloc(sz * sizeof(T));
  hipMemcpy(host_arr, devicebuff, sz*sizeof(T), hipMemcpyDeviceToHost);
  printf("norm arr %f", host_arr[0]);
  /* } */
}

// reduce (sum) a vector over two devices.
template <typename T>
void reduce_sendbufftor(){
  int nDev = 2;
  int sz = 32;

  int devs[2] = {0, 1};
  T value = 10.;
  T* host_v = (T*)malloc(sz * sizeof(T));
  for (int i = 0; i < sz; i++){
    host_v[i] = value;
  }

  T** sendbuff = (T**)malloc(nDev * sizeof(T*));
  T** recvbuff = (T**)malloc(nDev * sizeof(T*));
  
  /* hipMalloc(&recvbuff, nDev * sizeof(T)); */
  /* hipMalloc(&reduced_result, nDev * sizeof(T)); */

  ncclComm_t comms[nDev];
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);

  for (int i = 0; i < nDev; ++i) {
    hipSetDevice(i);
    hipMalloc(sendbuff + i, sz * sizeof(T));
    hipMalloc(recvbuff + i, sizeof(T));
    hipMemcpy(sendbuff[i], host_v, sz * sizeof(T), hipMemcpyHostToDevice); /* T float_val = 1.0; */ 
    hipStreamCreate(s+i);
  }

  check_arrays(sz, sendbuff);
  ncclCommInitAll(comms, nDev, devs);
  ncclGroupStart();
  for (int i = 0; i < nDev; ++i)
    ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], sz, ncclFloat, ncclSum, comms[i], s[i]);
  ncclGroupEnd();
  
  //synchronizing on CUDA streams to wait for completion of NCCL operation
  for (int i = 0; i < nDev; ++i) {
    hipSetDevice(i);
    hipStreamSynchronize(s[i]);
  }

  T val = -999.;
  T* mem = (T*)malloc(nDev * sizeof(T));
  hipMemcpy(&val, &recvbuff[0][0], sizeof(T), hipMemcpyDeviceToHost);
  printf("single value %f\n", val);
}


// based on example 1: single_process_multiple_devices_nccl.c
// https://docs.nvidia.com/deeplearning/nccl/user-guide/docs/examples.html
//  Single Process, Single Thread, Multiple Devices
template <typename T>
int vector_norm_ssm(){
  int nDev = 2;
  int devs[2] = {0, 1};
  // vector length
  int size = 32;

  // initialize array on host
  T value = 10.;
  T* host_v = (T*)malloc(size * sizeof(T));
  for (int i = 0; i < size; i++){
    host_v[i] = value;
  }

  //allocating and initializing device buffers
  T** vec = (T**)malloc(nDev * sizeof(T*));
  T* dot_result = (T*)malloc(nDev * sizeof(T));
  T* reduced_result = (T*)malloc(nDev * sizeof(T));

  hipMalloc(&reduced_result, nDev * sizeof(T));
  hipMalloc(&dot_result, nDev * sizeof(T));

  ncclComm_t comms[nDev];
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);
  
  for (int i = 0; i < nDev; ++i) {
    hipSetDevice(i);
    hipMalloc(vec + i, size * sizeof(T));
    hipMemcpy(vec[i], host_v, size * sizeof(T), hipMemcpyHostToDevice);
    hipStreamCreate(s+i);
  }
  check_arrays(size, vec);

  
  //initializing NCCL
  ncclCommInitAll(comms, nDev, devs);

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle,HIPBLAS_POINTER_MODE_DEVICE); // set here!!!

  
  //calling NCCL communication API. Group API is required when using
  //multiple devices per thread

  T* result = (T*)malloc(sizeof(T));

  ncclGroupStart();
  for (int i = 0; i < nDev; ++i){
    // dot product kernel
    /* dot(handle, size * sizeof(T), vec[i], vec[i]); */
    /* dot<<<1,16>>>(size,  vec[i], vec[i], &recvbuff[i][0]); */
    /* dot<<<1,1>>>(size,  vec[i], vec[i], &dot_result[i]); */
    /* full_dot<<<1,1>>>(vec[i], vec[i], &dot_result[i], size); */

    // ncclAllReduce(vec, recvbuff, num_of_elements, type, reduction_operation, comms, streams)

    hipblasSdot(handle, size, vec[i], 1, vec[i], 1, &dot_result[i]);
    /* hipblasSdot(handle, size, vec[i], 1, vec[i], 1, result); */
    ncclAllReduce((const void*)&dot_result[i], (void*)&reduced_result[i], 1, ncclFloat,\
        ncclSum, comms[i], s[i]);
  }
  ncclGroupEnd();


 
  //synchronizing on CUDA streams to wait for completion of NCCL operation
  for (int i = 0; i < nDev; ++i) {
    hipSetDevice(i);
    hipStreamSynchronize(s[i]);
  }


  // check the value of the AllReduce  
  T val = -999.;
  hipMemcpy(&val, &reduced_result[1], sizeof(T), hipMemcpyDeviceToHost);
  printf("dot %f \n", val);
  printf("%f\n", result);
 

  hipblasDestroy(handle);
  //free device buffers
  for (int i = 0; i < nDev; ++i) {
    hipSetDevice(i);
    hipFree(vec[i]);
    hipFree(&dot_result[i]);
    hipFree(&reduced_result[i]);
    /* hipFree(recvbuff[i]); */
  }


  //finalizing NCCL
  for (int i = 0; i < nDev; ++i) 
    ncclCommDestroy(comms[i]);
  
  return 0;
}

int main()
{
  vector_norm_ssm<float>();
}
